#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "common.h"

const int BlockSize = 256;

__global__ void dot(float *a, float *b, float *c, int N) {
   /* Use cache to store the products of each index of the arrays in a locally shared array for each block */
   __shared__ float cache[BlockSize];
   /* Calculate thread id (according to slide 24) */
   int id = blockDim.x * blockIdx.x + threadIdx.x;

   cache[threadIdx.x] = a[id] * b[id];
   __syncthreads();

   /* A recursive approach to do the additions in parallel */
   for (int i = blockDim.x/2; i >  0; i /= 2) {
      if (threadIdx.x < i) {
         cache[threadIdx.x] += cache[threadIdx.x + i];
      }
      __syncthreads();
   }

   /* Store the result from this block in global memory */
   if (threadIdx.x == 0) {
      c[blockIdx.x] = cache[0];
   }

}

int main(int argc, char* argv[])
{
  if (argc != 3) {
    fprintf(stderr, "Syntax: %s <vector size N> <device id>\n", argv[0]);
    return EXIT_FAILURE;
  }

  int N = atoi(argv[1]);
  int GridSize = imin(pow(2, 31), (N + BlockSize - 1) / BlockSize);
  int devId = atoi(argv[2]);
  struct timespec start, finish, partial_start, partial_finish;
  double elapsed;

#if defined(DEBUG) || defined(_DEBUG)
  printf("DeviceId=%d, N=%d, BlockSize=%d, GridSize=%d\n", devId, N, BlockSize, GridSize);
#endif

  checkCuda(hipSetDevice(devId));

  float *a, *b, c, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;

  // allocate memory on the cpu side
  a = (float*) malloc(N * sizeof(float));
  b = (float*) malloc(N * sizeof(float));
  partial_c = (float*) malloc(GridSize * sizeof(float));

  assert(a != NULL);
  assert(b != NULL);
  assert(partial_c != NULL);

  // allocate the memory on the GPU
  checkCuda(hipMalloc((void**) &dev_a, N * sizeof(float)));
  checkCuda(hipMalloc((void**) &dev_b, N * sizeof(float)));
  checkCuda(hipMalloc((void**) &dev_partial_c, GridSize * sizeof(float)));

  // fill in the host memory with data
  for (int i = 0; i < N; ++i) {
    a[i] = i;
    b[i] = i * 2;
  }

  clock_gettime(CLOCK_MONOTONIC_RAW, &start);
  partial_start = start;
  // copy the arrays 'a' and 'b' to the GPU
  checkCuda(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));

  clock_gettime(CLOCK_MONOTONIC_RAW, &partial_finish);
  elapsed = calculate_elapsed_time(partial_start, partial_finish);
  printf("Memcpy 1 elapsed time: %lf\n", elapsed);

  clock_gettime(CLOCK_MONOTONIC_RAW, &partial_start);
  dot<<<GridSize,BlockSize>>>(dev_a, dev_b, dev_partial_c, N);
  clock_gettime(CLOCK_MONOTONIC_RAW, &partial_finish);
  elapsed = calculate_elapsed_time(partial_start, partial_finish);
  printf("Kernel elapsed time: %lf\n", elapsed);

  // copy the array 'c' back from the GPU to the CPU
  clock_gettime(CLOCK_MONOTONIC_RAW, &partial_start);
  checkCuda(hipMemcpy(partial_c, dev_partial_c, GridSize * sizeof(float), hipMemcpyDeviceToHost));
  clock_gettime(CLOCK_MONOTONIC_RAW, &partial_finish);
  elapsed = calculate_elapsed_time(partial_start, partial_finish);
  printf("Memcpy 2 elapsed time: %lf\n", elapsed);

  // finish up on the CPU side
  clock_gettime(CLOCK_MONOTONIC_RAW, &partial_start);
  c = 0;
  for (int i = 0; i < GridSize; ++i) {
    c += partial_c[i];
  }
  clock_gettime(CLOCK_MONOTONIC_RAW, &partial_finish);
  elapsed = calculate_elapsed_time(partial_start, partial_finish);
  printf("CPU-side loop elapsed time: %lf\n", elapsed);

  clock_gettime(CLOCK_MONOTONIC_RAW, &finish);
  elapsed = calculate_elapsed_time(start, finish);

  printf("Does GPU value %.6g = %.6g?\n", c, 2 * sum_squares((float)(N - 1)));
  printf("Total elapsed time: %lf\n", elapsed);

  // free memory on the gpu side
  checkCuda(hipFree(dev_a));
  checkCuda(hipFree(dev_b));
  checkCuda(hipFree(dev_partial_c));

  // free memory on the cpu side
  free(a);
  free(b);
  free(partial_c);
}
