#include "hip/hip_runtime.h"
#include <assert.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "common.h"

// See more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/#sthash.7QeV8b7J.dpuf

// Forward declaration of the matrix sum kernel
__global__ void MatSumKernel(const Matrix, const Matrix, Matrix);

// Matrix sum - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatSum(const Matrix A, const Matrix B, Matrix C)
{
  Matrix d_A, d_B, d_C;

  // Load A and B to device memory
  d_A.width = A.width; d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(double);
  checkCuda(hipMalloc(&d_A.elements, size));
  checkCuda(hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice));

  d_B.width = B.width; d_B.height = B.height;
  size = B.width * B.height * sizeof(double);
  checkCuda(hipMalloc(&d_B.elements, size));
  checkCuda(hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice));

  // Allocate C in device memory
  d_C.width = C.width; d_C.height = C.height;
  size = C.width * C.height * sizeof(double);
  hipMalloc(&d_C.elements, size);

  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
  MatSumKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

  // Read C from device memory
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}

// Matrix sum kernel called by MatSum()
__global__ void MatSumKernel(Matrix A, Matrix B, Matrix C)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  C.elements[row * C.width + col] =
    A.elements[row * A.width + col] + B.elements[row * B.width + col];
}

int main(int argc, char* argv[])
{
  if (argc != 3) {
    fprintf(stderr, "Syntax: %s <vector size N> <device id>\n", argv[0]);
    return EXIT_FAILURE;
  }

  const int N = atoi(argv[1]);
  const int devId = atoi(argv[2]);
  size_t size = N * N * sizeof(double);
  struct timespec start, finish;
  double elapsed;
  double mul = 5.0;

  checkCuda(hipSetDevice(devId));

  Matrix a, b, c;

  // allocate matrices on the CPU side
  a.width = N;
  a.height = N;
  a.elements = (double *) malloc(size);

  b.width = N;
  b.height = N;
  b.elements = (double *) malloc(size);

  c.width = N;
  c.height = N;
  c.elements = (double *) malloc(size);

  // fill in the host memory with data
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      a.elements[i * N + j] = i * mul;
      b.elements[i * N + j] = i;
    }
  }

  clock_gettime(CLOCK_MONOTONIC_RAW, &start);

  MatSum(a, b, c);

  clock_gettime(CLOCK_MONOTONIC_RAW, &finish);
  elapsed = calculate_elapsed_time(start, finish);

  printf("Total elapsed time: %lf\n", elapsed);

  // finish up on the CPU side
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      assert(compare_doubles(
             c.elements[i * N + j],
             a.elements[i * N + j] + b.elements[i * N + j],
             0.1));
    }
  }
  printf("Matrix check successful!\n");

  // free memory on the cpu side
  free(a.elements);
  free(b.elements);
  free(c.elements);
}
